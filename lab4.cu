#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

using namespace std;

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
			    }														\
} while (0)

__device__ __constant__ double avg_const[96];

__global__ void kernel_main(int height, int width, int nc, unsigned int *src)
{
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
	while (tid_y < height)
	{
		int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
		while (tid_x < width)
		{
			double p[3];
			double max = 0;
			int jc = 0;
			p[0] = src[tid_y * width + tid_x] & 0xFF;
			p[1] = (src[tid_y * width + tid_x] & 0xFF00) >> 8;
			p[2] = (src[tid_y * width + tid_x] & 0xFF0000) >> 16;
			for (int k = 0; k < nc; k++)
			{
				double arg = p[0] * avg_const[k * 3] + p[1] * avg_const[k * 3 + 1] + p[2] * avg_const[k * 3 + 2];
				if (arg > max)
				{
					max = arg;
					jc = k;
				}
			}
			src[tid_y * width + tid_x] |= jc << 24;
			tid_x += blockDim.x * gridDim.x;
		}
		tid_y += blockDim.y * gridDim.y;
	}
}

int main()
{
	string path_in, path_out;

	cin >> path_in >> path_out;

	int nc;
	cin >> nc;

	int width, height;
	FILE *in = fopen(path_in.c_str(), "rb");
	if (in == NULL)
	{
		cout << "ERROR: Incorrect input file.\n";
		return 0;
	}
	fread(&width, sizeof(int), 1, in);
	fread(&height, sizeof(int), 1, in);

	if (width <= 0 || height <= 0 || nc < 0 || nc > 32)
	{
		cout << "ERROR: Incorrect data.\n";
		return 0;
	}

	unsigned int *src = (unsigned int *)malloc(sizeof(unsigned int) * width * height);
	fread(src, sizeof(unsigned int), width * height, in);
	fclose(in);

	double avg[96] = { 0, };

	for (int i = 0; i < nc; i++)
	{
		double np;
		cin >> np;
		for (int j = 0; j < np; j++)
		{
			int x, y;
			cin >> x >> y;
			avg[i * 3]		+= src[y * width + x] & 0xFF;
			avg[i * 3 + 1]	+= (src[y * width + x] & 0xFF00) >> 8;
			avg[i * 3 + 2]	+= (src[y * width + x] & 0xFF0000) >> 16;
		}
		avg[i * 3] /= np;
		avg[i * 3 + 1] /= np;
		avg[i * 3 + 2] /= np;
		double modulus = sqrt(avg[i * 3] * avg[i * 3] + avg[i * 3 + 1] * avg[i * 3 + 1] + avg[i * 3 + 2] * avg[i * 3 + 2]);
		avg[i * 3] /= modulus;
		avg[i * 3 + 1] /= modulus;
		avg[i * 3 + 2] /= modulus;
	}

	CSC(hipMemcpyToSymbol(HIP_SYMBOL(avg_const), avg, sizeof(double) * 96));

	unsigned int *src_dev;
	CSC(hipMalloc(&src_dev, sizeof(unsigned int) * height * width));
	CSC(hipMemcpy(src_dev, src, sizeof(unsigned int) * height * width, hipMemcpyHostToDevice));

	dim3 threads_count(16, 16);
	dim3 blocks_count(16, 16);

	kernel_main << < blocks_count, threads_count >> >(height, width, nc, src_dev);

	CSC(hipMemcpy(src, src_dev, sizeof(unsigned int) * height * width, hipMemcpyDeviceToHost));
	CSC(hipFree(src_dev));

	FILE *out = fopen(path_out.c_str(), "wb");
	if (out == NULL)
	{
		cout << "ERROR: Incorrect output file.\n";
		return 0;
	}
	fwrite(&width, sizeof(int), 1, out);
	fwrite(&height, sizeof(int), 1, out);
	fwrite(src, sizeof(unsigned int), height * width, out);
	fclose(out);

	free(src);
	return 0;
}
